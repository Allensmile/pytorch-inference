#include "hip/hip_runtime.h"
//#include "pytorch_softmax.hpp"

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// CUDA: grid stride looping
#define CUDA_KERNEL_LOOP(i,n)                      \
for(int i = blockIdx.x * blockDim.x + threadIdx.x; \
i < (n);                                           \
i +=blockDim.x * gridDim.x)

// CUDA: use 512 threads per block
const int CAFFE_CUDA_NUM_THREADS = 512;

// CUDA: number of blocks for threads.
inline int CAFFE_GET_BLOCKS(const int N) { \
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}
  
//template <typename Dtype>
__global__ void softmax_exp0(float *a, int n, float *y) 
{
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = exp(a[i]);
  }
}

const int blockSize = 1024;
__global__ void softmax_sum(const float *gArr, int arraySize, float *gOut) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    float sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ float shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = log(shArr[0]);
}

template <typename Dtype>
__global__ void softmax_exp1(Dtype *a, int n, Dtype *b, Dtype *y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = exp(a[i] - b[0]);
  }
}

void pytorch_gpu_softmax(float *array, int size){
	float *gpudata, *y, *b;
	
	hipMalloc((void**)&gpudata, sizeof(float) * size);
    hipMalloc((void**)&y, sizeof(float) * size);
	hipMalloc((void**)&b, sizeof(float));
	
	hipMemset(gpudata, 0, sizeof(float) * size);
	hipMemset(y, 0, sizeof(float) * size);
	hipMemset(b, 0, sizeof(float));
	
	hipMemcpy(gpudata, array, sizeof(float) * size, hipMemcpyHostToDevice);
	
	softmax_exp0 << < CAFFE_GET_BLOCKS(1), CAFFE_CUDA_NUM_THREADS >> > (gpudata, size, y);
	softmax_sum << < CAFFE_GET_BLOCKS(1), CAFFE_CUDA_NUM_THREADS >> > (y, size, b);
	softmax_exp1 << < CAFFE_GET_BLOCKS(1), CAFFE_CUDA_NUM_THREADS >> > (gpudata, size, b, y);
	
	hipMemcpy(array, y, sizeof(float) * size, hipMemcpyDeviceToHost);
	
	hipFree(gpudata);
    hipFree(y);
	hipFree(b);
	
}

int main(int argc, char **argv) 
{
	std::vector<float> uniform;
    for (int i = 0; i < 3; i++) {
        uniform.push_back(float(i + 0.1f));
		std::cout << uniform[i] << std::endl;
    }
	std::cout << std::endl;

	pytorch_gpu_softmax(uniform.data(), 3);
	
	for (int i = 0; i < 3; i++) {
		std::cout << uniform[i] << std::endl;
	}

	return 0;
}